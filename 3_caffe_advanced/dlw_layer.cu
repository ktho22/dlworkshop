#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/neuron_layer.hpp"
#include "caffe/layers/dlw_layer.hpp"

namespace caffe {

// CUDA kernele for forward
template <typename Dtype>
__global__ void DlwForward(const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out, const Dtype* slope_data, const Dtype* bias_data, const Dtype* slope2_data,
    const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out[index] = in[index] > 0 ? in[index] : in[index] > bias_data[c] ? in[index] * slope_data[c] : in[index] * slope2_data[c]; //change
  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void DlwBackward(const int n, const int channels, const int dim,
    const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff,
    const Dtype* slope_data, const Dtype* bias_data, const Dtype* slope2_data, const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    if ( in_data[index] > bias_data[c] ) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * slope_data[c]);
    } else {
    out_diff[index] = in_diff[index] * slope2_data[c];  //change
    }
  }
}

// CUDA kernel for element-wise parameter backward
template <typename Dtype>
__global__ void DlwParamBackward(const int n,
    const int rows, const int rowPitch, const Dtype* in_diff,
    const Dtype* in_data, Dtype* slope_diff, Dtype* bias_diff, Dtype* slope2_diff, const Dtype* bias_data, const int channels, const int dim) {    //change
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels;
    slope_diff[index] = in_diff[index] * in_data[index] * (in_data[index] <= 0) * (in_data[index] >= bias_data[c]);
    bias_diff[index] = 0;
    slope2_diff[index] = in_diff[index] * in_data[index] * (in_data[index] <= bias_data[c]);
    for ( int k = 1; k < rows; k++ ) {
        slope_diff[index] += in_diff[index + k*rowPitch]
           * in_data[index + k*rowPitch] * (in_data[index + k*rowPitch] <= 0) * (in_data[index + k*rowPitch] >= bias_data[c]);
        slope2_diff[index] += in_diff[index + k*rowPitch]
           * in_data[index + k*rowPitch] * (in_data[index + k*rowPitch] <= bias_data[c]);    //change
    }
  }
}

template <typename Dtype>
void DlwLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();
  const Dtype* slope_data = this->blobs_[0]->gpu_data();
  const Dtype* bias_data = this->blobs_[1]->gpu_data();
  const Dtype* slope2_data = this->blobs_[2]->gpu_data(); //attach
  const int div_factor = 1;

  // For in-place computation
  if (top[0] == bottom[0]) {
    caffe_copy(count, bottom_data, bottom_memory_.mutable_gpu_data());
  }

  // NOLINT_NEXT_LINE(whitespace/operators)
  DlwForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, channels, dim, bottom_data, top_data, slope_data, bias_data, slope2_data, div_factor);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void DlwLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();

  // For in-place computation
  if (top[0] == bottom[0]) {
    bottom_data = bottom_memory_.gpu_data();
  }

  // Propagate to param
  // Since to write bottom diff will affect top diff if top and bottom blobs
  // are identical (in-place computaion), we first compute param backward to
  // keep top_diff unchanged.
  if (this->param_propagate_down_[0]) {
    Dtype* slope_diff = this->blobs_[0]->mutable_gpu_diff();
    Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
    Dtype* slope2_diff = this->blobs_[2]->mutable_gpu_diff();
    int cdim = channels * dim;

    // compute element-wise diff
    // NOLINT_NEXT_LINE(whitespace/operators)
    DlwParamBackward<Dtype><<<CAFFE_GET_BLOCKS(cdim),
      CAFFE_CUDA_NUM_THREADS>>>(
      cdim, bottom[0]->num(), top[0]->offset(1), top_diff ,
      bottom_data ,
      backward_slope_buff_.mutable_gpu_diff(), backward_bias_buff_.mutable_gpu_diff(), backward_slope2_buff_.mutable_gpu_diff(), this->blobs_[1]->gpu_data(), dim, channels);

    CUDA_POST_KERNEL_CHECK;

    caffe_gpu_gemv<Dtype>(CblasNoTrans, channels, dim, 1.,
        backward_slope_buff_.gpu_diff(), multiplier_.gpu_data(), 1.,
        slope_diff);
    caffe_gpu_gemv<Dtype>(CblasNoTrans, channels, dim, 1.,
        backward_bias_buff_.gpu_diff(), multiplier_.gpu_data(), 1.,
        bias_diff);
    caffe_gpu_gemv<Dtype>(CblasNoTrans, channels, dim, 1.,
        backward_slope2_buff_.gpu_diff(), multiplier_.gpu_data(), 1.,
        slope2_diff);
    
  }
  // Propagate to bottom
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* slope_data = this->blobs_[0]->gpu_data();
    const Dtype* bias_data = this->blobs_[1]->gpu_data();
    const Dtype* slope2_data = this->blobs_[2]->gpu_data();
    int div_factor = channel_shared_ ? channels : 1;
    // NOLINT_NEXT_LINE(whitespace/operators)
    DlwBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
        count, channels, dim, top_diff, bottom_data, bottom_diff, slope_data, bias_data, slope2_data,
        div_factor);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(DlwLayer);


}  // namespace caffe
